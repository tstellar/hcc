
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple nvptx64-unknown-unknown | FileCheck -check-prefix=NVPTX %s
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple amdgcn-amd-amdhsa | FileCheck -check-prefix=AMDGCN %s


// Make sure we emit the proper addrspacecast for llvm.used.  PR22383 exposed an
// issue where we were generating a bitcast instead of an addrspacecast.

// NVPTX: @llvm.used = appending global [1 x i8*] [i8* addrspacecast (i8 addrspace(1)* bitcast ([0 x i32] addrspace(1)* @a to i8 addrspace(1)*) to i8*)], section "llvm.metadata"
// AMDGCN: @llvm.used = appending global [1 x i8 addrspace(4)*] [i8 addrspace(4)* addrspacecast (i8 addrspace(1)* bitcast ([0 x i32] addrspace(1)* @a to i8 addrspace(1)*) to i8 addrspace(4)*)], section "llvm.metadata"
__attribute__((device)) __attribute__((__used__)) int a[] = {};
